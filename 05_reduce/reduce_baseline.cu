/**
 * reduce_sum
 * baseline: 串行处理
 * latency: 1027.339844 ms
 */
#include <hip/hip_runtime.h>
#include <iostream>

// 错误检查宏
#define cudaCheckError() {                                      \
    hipError_t e = hipGetLastError();                         \
    if (e != hipSuccess) {                                     \
        printf("CUDA Error %s:%d: %s\n", __FILE__, __LINE__,    \
                hipGetErrorString(e));                         \
        exit(EXIT_FAILURE);                                     \
    }                                                           \
}

__global__ void reduce_baseline(const int* input, int* output, size_t n) {
    // 由于只分配了1个block和thread,此时cuda程序相当于串行程序
    int sum = 0;
    for(size_t i = 0; i < n; ++i) {
        sum += input[i];
    }
    *output = sum;
}

bool checkResult(int* out, int groudtruth, int n) {
    if (*out != groudtruth) {
        return false;
    }
    return true;
}

int main() {
    // const int N = 32 * 1024 * 1024;
    const int N = 25600000;

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const int blockSize = 1;
    int gridSize = 1;

    int *a = (int *)malloc(N * sizeof(int));
    int *d_a;
    hipMalloc((void **)&d_a, N * sizeof(int));

    int *out = (int *)malloc(gridSize * sizeof(int));  // 为啥这里是gridSize(每个block内的线程reduce到一起吗)
    int *d_out;
    hipMalloc((void **)&d_out, gridSize * sizeof(int));

    for(int i = 0; i < N; i++) {
        a[i] = 1;
    }
    int groudtruth = N * 1;
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);

    // 定义block数量和threads的数量
    dim3 Grid(gridSize);
    dim3 Block(blockSize);

    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_baseline<<<1, 1>>>(d_a, d_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    cudaCheckError();
    hipMemcpy(out, d_out, gridSize * sizeof(int), hipMemcpyDeviceToHost);
    bool is_right = checkResult(out, groudtruth, gridSize);
    
    if(is_right) {
        printf("the ans is right!\n");
    }
    else {
        printf("the ans is wrong!\n");
        for(int i = 0; i < gridSize;i++){
            printf("res per block : %1f ",out[i]);
        }
        printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_baseline latency = %f ms\n", milliseconds);


    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}